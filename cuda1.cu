#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>

using namespace std;

// Constants are the integer part of the sines of integers (in radians) * 2^32.

__device__ const uint32_t k[64] = {

0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee ,

0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501 ,

0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be ,

0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821 ,

0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa ,

0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8 ,

0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed ,

0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a ,

0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c ,

0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70 ,

0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05 ,

0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665 ,

0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039 ,

0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1 ,

0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1 ,

0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 };

// r specifies the per-round shift amounts

__device__ const uint32_t r[] = { 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,

                      5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20,

                      4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,

                      6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21 };

// leftrotate function definition

#define LEFTROTATE(x, c) (((x) << (c)) | ((x) >> (32 - (c))))

__device__ void to_bytes(uint32_t val, uint8_t* bytes)

{

    bytes[0] = (uint8_t)val;

    bytes[1] = (uint8_t)(val >> 8);

    bytes[2] = (uint8_t)(val >> 16);

    bytes[3] = (uint8_t)(val >> 24);

}

__device__ uint32_t to_int32(const uint8_t* bytes)

{

    return (uint32_t)bytes[0]

        | ((uint32_t)bytes[1] << 8)

        | ((uint32_t)bytes[2] << 16)

        | ((uint32_t)bytes[3] << 24);

}

__device__ void md5(const uint8_t* initial_msg, size_t initial_len, uint8_t* digest) {

    // These vars will contain the hash

    uint32_t h0, h1, h2, h3;

    // Message (to prepare)

    uint8_t* msg = NULL;

    size_t new_len, offset;

    uint32_t w[16];

    uint32_t a, b, c, d, i, f, g, temp;

    // Initialize variables - simple count in nibbles:

    h0 = 0x67452301;

    h1 = 0xefcdab89;

    h2 = 0x98badcfe;

    h3 = 0x10325476;

    //Pre-processing:

    //append "1" bit to message    

    //append "0" bits until message length in bits ≡ 448 (mod 512)

    //append length mod (2^64) to message

    for (new_len = initial_len + 1; new_len % (512 / 8) != 448 / 8; new_len++)

        ;

    msg = (uint8_t*)malloc(new_len + 8);

    memcpy(msg, initial_msg, initial_len);

    msg[initial_len] = 0x80; // append the "1" bit; most significant bit is "first"

    for (offset = initial_len + 1; offset < new_len; offset++)

        msg[offset] = 0; // append "0" bits

    // append the len in bits at the end of the buffer.

    to_bytes(initial_len * 8, msg + new_len);

    // initial_len>>29 == initial_len*8>>32, but avoids overflow.

    to_bytes(initial_len >> 29, msg + new_len + 4);

    // Process the message in successive 512-bit chunks:

    //for each 512-bit chunk of message:

    for (offset = 0; offset < new_len; offset += (512 / 8)) {

        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15

        for (i = 0; i < 16; i++)

            w[i] = to_int32(msg + offset + i * 4);

        // Initialize hash value for this chunk:

        a = h0;

        b = h1;

        c = h2;

        d = h3;

        // Main loop:

        for (i = 0; i < 64; i++) {

            if (i < 16) {

                f = (b & c) | ((~b) & d);

                g = i;

            }

            else if (i < 32) {

                f = (d & b) | ((~d) & c);

                g = (5 * i + 1) % 16;

            }

            else if (i < 48) {

                f = b ^ c ^ d;

                g = (3 * i + 5) % 16;

            }

            else {

                f = c ^ (b | (~d));

                g = (7 * i) % 16;

            }

            temp = d;

            d = c;

            c = b;

            b = b + LEFTROTATE((a + f + k[i] + w[g]), r[i]);

            a = temp;

        }

        // Add this chunk's hash to result so far:

        h0 += a;

        h1 += b;

        h2 += c;

        h3 += d;

    }

    // cleanup

    free(msg);

    //var char digest[16] := h0 append h1 append h2 append h3 //(Output is in little-endian)

    to_bytes(h0, digest);

    to_bytes(h1, digest + 4);

    to_bytes(h2, digest + 8);

    to_bytes(h3, digest + 12);

}

/* A utility function to reverse a string  */

__device__ void reverse(char str[], int length)

{

    int start = 0;

    int end = length - 1;

    while (start < end)

    {

        //swap(*(str + start), *(str + end));
        char tmp = *(str + start);
        *(str + start) = *(str + end);
        *(str + end) = tmp;

        start++;

        end--;

    }

}

// Implementation of itoa()

__device__ int itoaa(unsigned int num, char* str, int base)

{

    int i = 0;

    bool isNegative = false;

    // Handle 0 explicitely, otherwise empty string is printed for 0 

    if (num == 0)

    {

        str[i++] = '0';

        str[i] = '\0';

        return i-1;

    }

    // In standard itoa(), negative numbers are handled only with 

    // base 10. Otherwise numbers are considered unsigned.

    if (num < 0 && base == 10)

    {

        isNegative = true;

        num = -num;

    }

    // Process individual digits

    while (num != 0)

    {

        int rem = num % base;

        str[i++] = (rem > 9) ? (rem - 10) + 'a' : rem + '0';

        num = num / base;

    }

    // If number is negative, append '-'

    if (isNegative)

        str[i++] = '-';

    str[i] = '\0'; // Append string terminator

    // Reverse the string

    reverse(str, i);

    return i;

}

__global__ void thread_hierarchy(int len, uint8_t* hash, bool* canRunCuda)
{
    if(*canRunCuda){
        //uint8_t* result = new uint8_t[16];
        uint8_t result[16];
        //int alphaLen = 26;
        char alphabet[] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};

        int charsSize = 36;
        char chars[36] = {'0','1','2','3','4','5','6','7','8','9','a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
        
        unsigned int i = (blockIdx.x * 60000) + (1024*(blockIdx.y) + threadIdx.y);
        
        char buffer[32];
        int arraySize = itoaa(i, buffer, 26);

        char res[32];
        int resLen = 0;

        while (resLen + arraySize < len){
            res[resLen] = alphabet[0];
            resLen++;
        }
        for(int i = 0; i < arraySize; i++){
            int indexOfChar = 0;
            for(int a = 0; a < charsSize; a++){
                if(chars[a] == buffer[i])
                    indexOfChar = a;
            }
            res[resLen] = alphabet[indexOfChar];
            resLen++;
        }

        md5(reinterpret_cast<const uint8_t*>(res), resLen, result);

        bool notHash = false;
        for(int i = 0; i <16;i++){
            if(result[i] != hash[i]){
                notHash = true;
            }
        }
        if(notHash == false){
            printf("Found!\n");
            printf("Word is %s\n",res);
            *canRunCuda = false;
        }
        free(result);
    }
}

void run_cuda(uint8_t* hash, int len, unsigned int p)
{
	hipError_t cerr;

    int gridSize = 0;
    int gridSizeX = 0;

    if(p > 1024){
		gridSize = ceil(p / 1024) + 1;
	}else{
		gridSize = 1;
	}

    if(gridSize > 60000){
        gridSizeX = ceil(gridSize / 60000) + 1;
        gridSize = 60000;
    }else{
        gridSizeX = 1;
    }

    printf("Grid x size: %d \n", gridSizeX);
    printf("Grid y size: %d \n", gridSize);

    bool *canRun = (bool*)malloc(sizeof(bool)*1);
    *canRun = true;

    uint8_t* differenceArray;
    bool* canRunCuda;

    hipMalloc((void**)&differenceArray, sizeof(uint8_t)*16);
    hipMalloc((void**)&canRunCuda, sizeof(bool)*1);
    hipMemcpy(differenceArray, hash, sizeof(uint8_t)*16, hipMemcpyHostToDevice);
    hipMemcpy(canRunCuda, canRun, sizeof(bool)*1, hipMemcpyHostToDevice);
         //          (dev ptr)  <--- (host ptr)
	
    auto begin = std::chrono::steady_clock::now();
	// Thread creation from selected kernel:
	// first parameter dim3 is grid dimension
	// second parameter dim3 is block dimension
    thread_hierarchy<<< dim3( gridSizeX, gridSize ), dim3( 1, 1024 )>>>(len, differenceArray, canRunCuda);

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Output from printf is in GPU memory. 
	// To get its contens it is necessary to synchronize device.

	hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();
    printf("%d mics\n", (int)chrono::duration_cast<chrono::microseconds>(end - begin).count());
    printf("%d ms\n", (int)chrono::duration_cast<chrono::milliseconds>(end - begin).count());
}
